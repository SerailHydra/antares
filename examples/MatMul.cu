///524288/float32/input0,524288/float32/input1:524288/float32/output0
// backend = c-cuda
// CONFIG: 
// COMPUTE_V1: - einstein_v2("output0[N] = input0[N] + input1[N]", input_dict={"input0": {"dtype": "float32", "shape": [1024 * 512]}, "input1": {"dtype": "float32", "shape": [1024 * 512]}})

#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <mma.h>

#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <string>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

const int input_size0 = 64 * 1024 * 1024;
const int input_size1 = 64 * 1024 * 1024;
const int output_size = 64 * 1024 * 1024;

#include "cu_helper.h"

int main(int argc, char *argv[])
{
	checkCudaErrors(hipInit(0));
	hipDevice_t device;
	checkCudaErrors(hipDeviceGet(&device, 0));
	hipCtx_t context;
	checkCudaErrors(hipCtxCreate(&context, hipDeviceScheduleAuto/*hipDeviceScheduleYield*/ | hipDeviceMapHost, device));

	hipModule_t cuModule_;
	hipFunction_t cuda_func_;
	//checkCudaErrors(hipModuleLoad(&cuModule_, "my_kernel.out"));
	//checkCudaErrors(hipModuleGetFunction(&cuda_func_, cuModule_, "template_op_kernel0"));

	int id = atoi(argv[1]);
	//int TB_size = atoi(argv[2]);
	float *Ah, *Bh;
	hipDeviceptr_t Ad, Bd, Cd;
	Ah = (float*)malloc(input_size0 * sizeof(float));
	Bh = (float*)malloc(input_size1 * sizeof(float));
	//Ch = (float*)malloc(SIZE * sizeof(float));

	//hipMalloc((void **)&Ad, SIZE * sizeof(float));
	//hipMalloc((void **)&Bd, SIZE * sizeof(float));
	//hipMalloc((void **)&Cd, SIZE * sizeof(float));

	checkCudaErrors(hipMalloc(&Ad, sizeof(float) * input_size0));
	checkCudaErrors(hipMalloc(&Bd, sizeof(float) * input_size1));
	checkCudaErrors(hipMalloc(&Cd, sizeof(float) * output_size));

	void* param[] = {&Ad, &Bd, &Cd};

	for (int i = 0; i < input_size0; ++ i)
		Ah[i] = rand();
	for (int i = 0; i < input_size1; ++ i)
		Bh[i] = rand();

	checkCudaErrors(hipMemcpyHtoD(Ad, Ah, input_size0 * sizeof(float)));
	checkCudaErrors(hipMemcpyHtoD(Bd, Bh, input_size1 * sizeof(float)));

	std::string path = "results/MatMul/" + std::to_string(id);
	std::string code_path = path + "/my_kernel.cc";
	std::string mod_path = path + "/my_kernel.out";
	checkCudaErrors(hipModuleLoad(&cuModule_, mod_path.c_str()));
	checkCudaErrors(hipModuleGetFunction(&cuda_func_, cuModule_, "template_op_kernel0"));
	auto fp = fopen(code_path.c_str(), "r");
	int TB_size, TB_count;
	while (!feof(fp))
	{
		char *line;
		line = (char*)malloc(1000 * sizeof(char));
	       	fgets(line, 1000, fp);
		std::string std_line = std::string(line);
		if (int(std_line.find("[thread_extent] blockIdx.x")) > -1)
		{
			int k = std_line.rfind("=");
			TB_count = std::atoi(std_line.substr(k + 2, std_line.length() - k).c_str());
		}
		if (int(std_line.find("[thread_extent] threadIdx.x")) > -1)
		{
			int k = std_line.rfind("=");
			TB_size = std::atoi(std_line.substr(k + 2, std_line.length() - k).c_str());
		}
	}
	for (int i = 0; i < 1; ++ i)
	{
		checkCudaErrors(hipModuleLaunchKernel(cuda_func_, TB_count, 1, 1, TB_size, 1, 1, 0, 0, (void**) param, 0));
		hipDeviceSynchronize();
	}

}

